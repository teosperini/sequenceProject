#include "hip/hip_runtime.h"
/*
 * Exact genetic sequence alignment
 * (Using brute force)
 *
 * CUDA version
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2023/2024
 *
 * v1.3
 *
 * (c) 2024, Arturo Gonzalez-Escribano
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<limits.h>
#include<sys/time.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>

/* Example of macros for error checking in CUDA */
#define CUDA_CHECK_FUNCTION( call )	{ hipError_t check = call; if ( check != hipSuccess ) fprintf(stderr, "CUDA Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }
#define CUDA_CHECK_KERNEL( )	{ hipError_t check = hipGetLastError(); if ( check != hipSuccess ) fprintf(stderr, "CUDA Kernel Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }

/* Arbitrary value to indicate that no matches are found */
#define	NOT_FOUND	-1

/* Arbitrary value to restrict the checksums period */
#define CHECKSUM_MAX	65535


/* 
 * Utils: Function to get wall time
 */
double cp_Wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}

/*
 * Utils: Random generator
 */
#include "rng.c"

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */
 // TODO INIZIO 
/* ADD KERNELS AND OTHER FUNCTIONS HERE */

/* 
 * KERNEL principale: scandisce la sequenza (a chunk) e controlla se il pattern corrisponde
 *
 * d_sequence:    sequenza globale su GPU
 * seq_length:    lunghezza della sequenza (intera)
 * seqStart:      punto di inizio del chunk da analizzare
 * chunkLen:      lunghezza del chunk in questa invocazione
 *
 * d_patterns:    array di puntatori ai pattern su GPU
 * d_pat_length:  array con le lunghezze di ciascun pattern
 * patStart e patEnd: intervallo dei pattern da processare in questo batch
 *
 * d_pat_found:   array (grande pat_number) dove salviamo la posizione di match 
 *                (inizialmente impostato a ULLONG_MAX se "non trovato").
 * d_pat_matches: contatore globale del numero di pattern trovati
 *
 * La griglia è configurata con dimensioni (gridX, gridY), dove:
 *    - offset (thread x) è l’offset sulla sequenza
 *    - localPat (thread y) è l’indice di pattern relativo nel batch 
 */
__global__ void matchPatternsKernel(
	char* d_sequence, unsigned long seq_length, unsigned long seqStart, unsigned long chunkLen,
	char** d_patterns, unsigned long* d_pat_length, unsigned long patStart, unsigned long patEnd,
	unsigned long long* d_pat_found, int* d_pat_matches)
{

	// Posizione della sequenza che stiamo analizzando
	unsigned long offset = blockIdx.x * blockDim.x + threadIdx.x;
	// Quale pattern stiamo controllando
	unsigned long localPat = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned long patIndex = patStart + localPat; // Indice assoluto del pattern
	unsigned long patLen = d_pat_length[patIndex]; // Lunghezza del pattern

	// Controllo di non andare oltre la lunghezza totale della sequenza
	if (seqStart + offset + patLen > seq_length) {
		return;
	}

	// Controllo di non sforare il chunk
	if (offset + patLen > chunkLen) {
		return;
	}

	// Confronto carattere per carattere
	bool match = true;
	for (unsigned long j = 0; j < patLen; j++) {
		// Se un carattere non corrisponde, il pattern non è presente in questa posizione
		if (d_sequence[seqStart + offset + j] != d_patterns[patIndex][j]) {
			match = false;
			break;
		}
	}

	__syncthreads();

	/*Se c'è corrispondenza: 
	 * - atomicCAS su d_pat_found, in modo che solo il primo thread che entra scrive la posizione (seqStart+offset)
	 * - atomicAdd su d_pat_matches per incrementare il contatore di pattern trovati
	 */
	if (match) {
		if (atomicCAS(&d_pat_found[patIndex], ULLONG_MAX, seqStart + offset) == ULLONG_MAX) {
			atomicAdd(d_pat_matches, 1);
		} else {
			atomicMin(&d_pat_found[patIndex], seqStart + offset);
		}
	}

}

/* 
 * KERNEL per incrementare i match sulla sequenza
 *	un thread per ciascun (patternId, offsetPattern)
 *
 * d_pat_found:   array con tutte le posizioni iniziali trovate (ULLONG_MAX se non trovato match)
 * d_pat_length: lunghezza di ciascun pattern
 * d_seq_matches: array di match sulla sequenza, quante volte ogni posizione della sequenza è stata coperta da uno o più pattern
 * pat_number: totale pattern
 * seq_length:    lunghezza della sequenza (intera)
 */
 __global__ void incrementMatchesKernel(
	const unsigned long long* d_pat_found,
	const unsigned long* d_pat_length,      
	int* d_seq_matches,                     
	int pat_number,
	unsigned long seq_length
) {
	// Quale pattern stiamo processando
	int patId = blockIdx.x * blockDim.x + threadIdx.x;
	// Offset del pattern (posizione della lettera all'interno del pattern)
	int localOffset = blockIdx.y * blockDim.y + threadIdx.y;

	// Fuori dal numero totale di pattern?
	if (patId >= pat_number) return;

	// Posizione iniziale pattern
	unsigned long long startPos = d_pat_found[patId];
	if (startPos == ULLONG_MAX) {
		// Pattern non trovato	
		return;
	}
	// Lunghezza del pattern
	unsigned long length = d_pat_length[patId];
	// Troppo grande? esco
	if (localOffset >= length) return;

	// Calcola la posizione esatta sulla sequenza
	unsigned long long pos = startPos + localOffset;
	// Controllo per non sforare la sequenza
	if (pos >= seq_length) return;

	// Incremento atomico del numero di match nella sequenza 
	atomicAdd(&d_seq_matches[pos], 1);
}


/*
 * Data una seq_length, restituisce la dimensione del chunk
 * da usare per la suddivisione della sequenza.
 */
unsigned long get_chunk_size(unsigned long seq_length) {
	if (seq_length < 1024) return seq_length;  // Se piccolo, nessuna suddivisione
	if (seq_length < 1000000) return max(1024UL, min(65536UL, (unsigned long)seq_length / 8));
	return max(65536UL, min(1048576UL, (unsigned long)seq_length / 10));
}

// TODO FINE
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

/*
 * Function: Allocate new patttern
 */
char *pattern_allocate( rng_t *random, unsigned long pat_rng_length_mean, unsigned long pat_rng_length_dev, unsigned long seq_length, unsigned long *new_length ) {

	/* Random length */
	unsigned long length = (unsigned long)rng_next_normal( random, (double)pat_rng_length_mean, (double)pat_rng_length_dev );
	if ( length > seq_length ) length = seq_length;
	if ( length <= 0 ) length = 1;

	/* Allocate pattern */
	char *pattern = (char *)malloc( sizeof(char) * length );
	if ( pattern == NULL ) {
		fprintf(stderr,"\n-- Error allocating a pattern of size: %lu\n", length );
		exit( EXIT_FAILURE );
	}

	/* Return results */
	*new_length = length;
	return pattern;
}

/*
 * Function: Fill random sequence or pattern
 */
void generate_rng_sequence( rng_t *random, float prob_G, float prob_C, float prob_A, char *seq, unsigned long length) {
	unsigned long ind;
	for( ind=0; ind<length; ind++ ) {
		double prob = rng_next( random );
		if( prob < prob_G ) seq[ind] = 'G';
		else if( prob < prob_C ) seq[ind] = 'C';
		else if( prob < prob_A ) seq[ind] = 'A';
		else seq[ind] = 'T';
	}
}

/*
 * Function: Copy a sample of the sequence
 */
void copy_sample_sequence( rng_t *random, char *sequence, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Copy sample */
	unsigned long ind;
	for( ind=0; ind<length; ind++ )
		pattern[ind] = sequence[ind+location];
}

/*
 * Function: Regenerate a sample of the sequence
 */
void generate_sample_sequence( rng_t *random, rng_t random_seq, float prob_G, float prob_C, float prob_A, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length ) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Regenerate sample */
	rng_t local_random = random_seq;
	rng_skip( &local_random, location );
	generate_rng_sequence( &local_random, prob_G, prob_C, prob_A, pattern, length);
}


/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s ", program_name );
	fprintf(stderr,"<seq_length> <prob_G> <prob_C> <prob_A> <pat_rng_num> <pat_rng_length_mean> <pat_rng_length_dev> <pat_samples_num> <pat_samp_length_mean> <pat_samp_length_dev> <pat_samp_loc_mean> <pat_samp_loc_dev> <pat_samp_mix:B[efore]|A[fter]|M[ixed]> <long_seed>\n");
	fprintf(stderr,"\n");
}



/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	/* 0. Default output and error without buffering, forces to write immediately */
	setbuf(stdout, NULL);
	setbuf(stderr, NULL);

	/* 1. Read scenary arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc < 15) {
		fprintf(stderr, "\n-- Error: Not enough arguments when reading configuration from the command line\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	/* 1.2. Read argument values */
	unsigned long seq_length = atol( argv[1] );
	float prob_G = atof( argv[2] );
	float prob_C = atof( argv[3] );
	float prob_A = atof( argv[4] );
	if ( prob_G + prob_C + prob_A > 1 ) {
		fprintf(stderr, "\n-- Error: The sum of G,C,A,T nucleotid probabilities cannot be higher than 1\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}
	prob_C += prob_G;
	prob_A += prob_C;

	int pat_rng_num = atoi( argv[5] );
	unsigned long pat_rng_length_mean = atol( argv[6] );
	unsigned long pat_rng_length_dev = atol( argv[7] );

	int pat_samp_num = atoi( argv[8] );
	unsigned long pat_samp_length_mean = atol( argv[9] );
	unsigned long pat_samp_length_dev = atol( argv[10] );
	unsigned long pat_samp_loc_mean = atol( argv[11] );
	unsigned long pat_samp_loc_dev = atol( argv[12] );

	char pat_samp_mix = argv[13][0];
	if ( pat_samp_mix != 'B' && pat_samp_mix != 'A' && pat_samp_mix != 'M' ) {
		fprintf(stderr, "\n-- Error: Incorrect first character of pat_samp_mix: %c\n\n", pat_samp_mix);
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	unsigned long seed = atol( argv[14] );

#ifdef DEBUG
	/* DEBUG: Print arguments */
	printf("\nArguments: seq_length=%lu\n", seq_length );
	printf("Arguments: Accumulated probabilitiy G=%f, C=%f, A=%f, T=1\n", prob_G, prob_C, prob_A );
	printf("Arguments: Random patterns number=%d, length_mean=%lu, length_dev=%lu\n", pat_rng_num, pat_rng_length_mean, pat_rng_length_dev );
	printf("Arguments: Sample patterns number=%d, length_mean=%lu, length_dev=%lu, loc_mean=%lu, loc_dev=%lu\n", pat_samp_num, pat_samp_length_mean, pat_samp_length_dev, pat_samp_loc_mean, pat_samp_loc_dev );
	printf("Arguments: Type of mix: %c, Random seed: %lu\n", pat_samp_mix, seed );
	printf("\n");
#endif // DEBUG

		CUDA_CHECK_FUNCTION( hipSetDevice(0) );

	/* 2. Initialize data structures */
	/* 2.1. Skip allocate and fill sequence */
	rng_t random = rng_new( seed );
	rng_skip( &random, seq_length );

	/* 2.2. Allocate and fill patterns */
	/* 2.2.1 Allocate main structures */
	int pat_number = pat_rng_num + pat_samp_num;
	unsigned long *pat_length = (unsigned long *)malloc( sizeof(unsigned long) * pat_number );
	char **pattern = (char **)malloc( sizeof(char*) * pat_number );
	if ( pattern == NULL || pat_length == NULL ) {
		fprintf(stderr,"\n-- Error allocating the basic patterns structures for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}

	/* 2.2.2 Allocate and initialize ancillary structure for pattern types */
	int ind;
	unsigned long lind;
	#define PAT_TYPE_NONE	0
	#define PAT_TYPE_RNG	1
	#define PAT_TYPE_SAMP	2
	char *pat_type = (char *)malloc( sizeof(char) * pat_number );
	if ( pat_type == NULL ) {
		fprintf(stderr,"\n-- Error allocating ancillary structure for pattern of size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_NONE;

	/* 2.2.3 Fill up pattern types using the chosen mode */
	switch( pat_samp_mix ) {
	case 'A':
		for( ind=0; ind<pat_rng_num; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		break;
	case 'B':
		for( ind=0; ind<pat_samp_num; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		break;
	default:
		if ( pat_rng_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		}
		else if ( pat_samp_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		}
		else if ( pat_rng_num < pat_samp_num ) {
			int interval = pat_number / pat_rng_num;
			for( ind=0; ind<pat_number; ind++ )
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_RNG;
				else pat_type[ind] = PAT_TYPE_SAMP;
		}
		else {
			int interval = pat_number / pat_samp_num;
			for( ind=0; ind<pat_number; ind++ )
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_SAMP;
				else pat_type[ind] = PAT_TYPE_RNG;
		}
	}

	/* 2.2.4 Generate the patterns */
	for( ind=0; ind<pat_number; ind++ ) {
		if ( pat_type[ind] == PAT_TYPE_RNG ) {
			pattern[ind] = pattern_allocate( &random, pat_rng_length_mean, pat_rng_length_dev, seq_length, &pat_length[ind] );
			generate_rng_sequence( &random, prob_G, prob_C, prob_A, pattern[ind], pat_length[ind] );
		}
		else if ( pat_type[ind] == PAT_TYPE_SAMP ) {
			pattern[ind] = pattern_allocate( &random, pat_samp_length_mean, pat_samp_length_dev, seq_length, &pat_length[ind] );
#define REGENERATE_SAMPLE_PATTERNS
#ifdef REGENERATE_SAMPLE_PATTERNS
			rng_t random_seq_orig = rng_new( seed );
			generate_sample_sequence( &random, random_seq_orig, prob_G, prob_C, prob_A, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#else
			copy_sample_sequence( &random, sequence, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#endif
		}
		else {
			fprintf(stderr,"\n-- Error internal: Paranoic check! A pattern without type at position %d\n", ind );
			exit( EXIT_FAILURE );
		}
	}
	free( pat_type );

	/* Allocate and move the patterns to the GPU */
	unsigned long *d_pat_length;
	char **d_pattern;
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pat_length, sizeof(unsigned long) * pat_number ) );
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pattern, sizeof(char *) * pat_number ) );

	char **d_pattern_in_host = (char **)malloc( sizeof(char*) * pat_number );
	if ( d_pattern_in_host == NULL ) {
		fprintf(stderr,"\n-- Error allocating the patterns structures replicated in the host for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) {
		CUDA_CHECK_FUNCTION( hipMalloc( &(d_pattern_in_host[ind]), sizeof(char) * pat_length[ind] ) );
		CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern_in_host[ind], pattern[ind], pat_length[ind] * sizeof(char), hipMemcpyHostToDevice ) );
	}
	CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern, d_pattern_in_host, pat_number * sizeof(char *), hipMemcpyHostToDevice ) );
	CUDA_CHECK_FUNCTION( hipMemcpy(d_pat_length, pat_length, sizeof(unsigned long) * pat_number, hipMemcpyHostToDevice) );


	/* Avoid the usage of arguments to take strategic decisions
	 * In a real case the user only has the patterns and sequence data to analize
	 */
	argc = 0;
	argv = NULL;
	pat_rng_num = 0;
	pat_rng_length_mean = 0;
	pat_rng_length_dev = 0;
	pat_samp_num = 0;
	pat_samp_length_mean = 0;
	pat_samp_length_dev = 0;
	pat_samp_loc_mean = 0;
	pat_samp_loc_dev = 0;
	pat_samp_mix = '0';

	/* 2.3. Other result data and structures */
	int pat_matches = 0;

	/* 2.3.1. Other results related to patterns */
	unsigned long long *pat_found;
	pat_found = (unsigned long long*)malloc( sizeof(unsigned long long) * pat_number );
	if ( pat_found == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux pattern structure for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}

	/* 3. Start global timer */
		CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */

	/* 4. Allocazione memoria per la sequenza su host*/
	char *sequence = (char *)malloc( sizeof(char) * seq_length );
	if ( sequence == NULL ) {
		fprintf(stderr,"\n-- Error allocating the sequence for size: %lu\n", seq_length );
		exit( EXIT_FAILURE );
	}

	/* 4.1 Generazione randomica dei caratteri della sequenza */
	random = rng_new( seed );
	generate_rng_sequence( &random, prob_G, prob_C, prob_A, sequence, seq_length);

	/* 4.2 Determinazione dimensione chunk e altri valori utili */
	unsigned long chunkSize = (unsigned long) get_chunk_size(seq_length);
	chunkSize = (chunkSize > seq_length) ? seq_length : chunkSize;

	unsigned long maxPatLength = 0;
	for (int i = 0; i < pat_number; i++) {
		if (pat_length[i] > maxPatLength) maxPatLength = pat_length[i];
	}
	// overlap = maxPatLength - 1 (se >0)
	unsigned long overlap = (maxPatLength > 0) ? (maxPatLength - 1) : 0;
	// Numero di pattern per batch
	unsigned long batchSize = 512;

	/* 4.3 Allocazione memoria e copia dati su GPU */
	char* d_sequence;
	int* d_pat_matches;
	int* d_seq_matches;
	unsigned long long* d_pat_found;

	CUDA_CHECK_FUNCTION(hipMalloc((void**)&d_sequence, seq_length * sizeof(char)));
	CUDA_CHECK_FUNCTION(hipMalloc((void**)&d_pat_matches, sizeof(int)));
	CUDA_CHECK_FUNCTION(hipMalloc((void**)&d_seq_matches, seq_length * sizeof(int)));
	CUDA_CHECK_FUNCTION(hipMalloc((void**)&d_pat_found, pat_number * sizeof(unsigned long long)));

	CUDA_CHECK_FUNCTION(hipMemcpy(d_sequence, sequence, seq_length * sizeof(char), hipMemcpyHostToDevice));
	CUDA_CHECK_FUNCTION(hipMemset(d_pat_matches, 0, sizeof(int)));

#ifdef DEBUG
	/* DEBUG: Print sequence and patterns */
	printf("-----------------\n");
	printf("Sequence: ");
	for( lind=0; lind<seq_length; lind++ )
		printf( "%c", sequence[lind] );
	printf("\n-----------------\n");
	printf("Patterns: %d ( rng: %d, samples: %d )\n", pat_number, pat_rng_num, pat_samp_num );
	int debug_pat;
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( "Pat[%d]: ", debug_pat );
		for( lind=0; lind<pat_length[debug_pat]; lind++ )
			printf( "%c", pattern[debug_pat][lind] );
		printf("\n");
	}
	printf("-----------------\n\n");
#endif // DEBUG

	/* 4.4 Allocazione array seq_matches su host */
	int* seq_matches;
	seq_matches = (int *)malloc( sizeof(int) * seq_length );
	if ( seq_matches == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux sequence structures for size: %lu\n", seq_length );
		exit( EXIT_FAILURE );
	}

	/* 4. Initialize ancillary structures */
	// Inizializza pat_found a NOT_FOUND, ma su device con ULLONG_MAX
	for( ind=0; ind<pat_number; ind++) {
		pat_found[ind] = (unsigned long long)NOT_FOUND;
	}
	// Inizializza seq_matches a NOT_FOUND, ma su device con 0
	for( lind=0; lind<seq_length; lind++) {
		seq_matches[lind] = NOT_FOUND;
	}
	CUDA_CHECK_FUNCTION(hipMemset(d_seq_matches, 0, seq_length * sizeof(int)));
	for (int i = 0; i < pat_number; i++) {
		pat_found[i] = ULLONG_MAX;
	}
	CUDA_CHECK_FUNCTION(hipMemcpy(d_pat_found, pat_found, pat_number * sizeof(unsigned long long), hipMemcpyHostToDevice));


	// Definizione della dimensione del blocco
	unsigned long dimBlockX = 32;
	unsigned long dimBlockY = 12;
	dim3 block(dimBlockX, dimBlockY);

	// Iterazione sui chunk di sequenza
	for (unsigned long seqStart = 0; seqStart < seq_length; seqStart += (chunkSize - overlap)) {
		unsigned long seqEnd = seqStart + chunkSize + overlap;
		seqEnd = (seqEnd > seq_length) ? seq_length : seqEnd;

		unsigned long chunkLen = seqEnd - seqStart;

		// Iterazione sui pattern in batch
		for (int patStart = 0; patStart < pat_number; patStart += batchSize) {
			// Ogni iterazione lavora su un batch di pattern
			int patEnd = patStart + batchSize;
			if (patEnd > pat_number) patEnd = pat_number;
			int numPatternsInThisBatch = patEnd - patStart;

			// Dimensioni griglia (gridX = posizioni sequenza, gridY = pattern)
			unsigned long gridX = (chunkLen + dimBlockX - 1) / dimBlockX;
			unsigned long gridY = (numPatternsInThisBatch + dimBlockY - 1) / dimBlockY;
			dim3 grid(gridX, gridY);

			// Lancio del kernel
			matchPatternsKernel<<<grid, block>>>(
				d_sequence,
				seq_length,
				seqStart,
				chunkLen,
				d_pattern,
				d_pat_length,
				patStart,
				patEnd,
				d_pat_found,
				d_pat_matches
			);
			CUDA_CHECK_KERNEL();
		}		
	}
	CUDA_CHECK_FUNCTION(hipDeviceSynchronize());
	// Copia di pat_matches da GPU a host
	CUDA_CHECK_FUNCTION(hipMemcpy(&pat_matches, d_pat_matches, sizeof(int), hipMemcpyDeviceToHost));
	dim3 blockIncrement(16, 16);
	/*
	* Dimensioni griglia 2D, in cui:
	*   - l'asse X copre i 'pat_number' pattern
	*   - l'asse Y copre la 'maxPatLength' (lunghezza massima pattern)
	*/
	dim3 grid(
		(pat_number + blockIncrement.x - 1) / blockIncrement.x,  // quante "righe" di blocchi per coprire tutti i pattern
		(maxPatLength + blockIncrement.y - 1) / blockIncrement.y   // quante "colonne" di blocchi per coprire la lunghezza massima
	);
	
	/* Questo kernel si occuperà di "incrementare" in parallelo i 
 	 * match sulla sequenza (d_seq_matches) per ogni 
 	 * (pattern, offset nel pattern)
	 */
	incrementMatchesKernel<<<grid, blockIncrement>>>(
		d_pat_found,
		d_pat_length,
		d_seq_matches,
		pat_number,
		seq_length
	);
	CUDA_CHECK_KERNEL();

	CUDA_CHECK_FUNCTION(hipDeviceSynchronize());
	// Copia di pat_found e seq_matches
	CUDA_CHECK_FUNCTION(hipMemcpy(pat_found, d_pat_found, pat_number * sizeof(unsigned long long), hipMemcpyDeviceToHost));
	// Sostituiamo ULLONG_MAX con NOT_FOUND
	for (int i = 0; i < pat_number; i++) {
		if (pat_found[i] == ULLONG_MAX) {
			pat_found[i] = (unsigned long long)NOT_FOUND;
		}
	}
	CUDA_CHECK_FUNCTION(hipMemcpy(seq_matches, d_seq_matches, seq_length * sizeof(int), hipMemcpyDeviceToHost));
	// Se > 0 (quindi se ha match), riduco di 1 (come in CPU)
	for (unsigned long i = 0; i < seq_length; i++) {
		if (seq_matches[i] > 0) {
			seq_matches[i]--;
		}else{
			seq_matches[i]=NOT_FOUND;
		}
	}	

	/* 7. Check sums */
	unsigned long long checksum_matches = 0;
	unsigned long checksum_found = 0;
	for( ind=0; ind < pat_number; ind++) {
		if ( pat_found[ind] != (unsigned long long)NOT_FOUND ){
			checksum_found = ( checksum_found + pat_found[ind] ) % CHECKSUM_MAX;
		}
	}
	for( lind=0; lind < seq_length; lind++) {
		if ( seq_matches[lind] != NOT_FOUND )
			checksum_matches = ( checksum_matches + seq_matches[lind] ) % CHECKSUM_MAX;
	}

#ifdef DEBUG
	/* DEBUG: Write results */
	printf("-----------------\n");
	printf("Found start:");
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( " %lu", pat_found[debug_pat] );
	}
	printf("\n");
	printf("-----------------\n");
	printf("Matches:");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( " %d", seq_matches[lind] );
	printf("\n");
	printf("-----------------\n");
#endif // DEBUG

	/* Free local resources */	
	free( sequence );
	free( seq_matches );

// TODO FINE 
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 8. Stop global timer */
		CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );
	ttotal = cp_Wtime() - ttotal;

	/* 9. Output for leaderboard */
	printf("\n");
	/* 9.1. Total computation time */
	printf("Time: %lf\n", ttotal );

	/* 9.2. Results: Statistics */
	printf("Result: %d, %lu, %llu\n\n", 
			pat_matches,
			checksum_found,
			checksum_matches );
		
	/* 10. Free resources */	
	int i;
	for( i=0; i<pat_number; i++ ) free( pattern[i] );
	free( pattern );
	free( pat_length );
	free( pat_found );
	hipFree(d_pattern);
	hipFree(d_pat_found);
	hipFree(d_pat_matches);
	hipFree(d_sequence);
	hipFree(d_pattern_in_host);
	hipFree(d_seq_matches);
	hipFree(d_pat_length);

	/* 11. End */
	return 0;
}
